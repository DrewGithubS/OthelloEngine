#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>

#include "GPUMethods.cudah"

const uint64_t ONE64GPU = 1;

const uint8_t shift1 = 1;
const uint8_t shift2 = 9;
const uint8_t shift3 = 8;
const uint8_t shift4 = 7;

const uint64_t MACROmask1 = 0x7F7F7F7F7F7F7F7F;
const uint64_t MACROmask2 = 0xFEFEFEFEFEFEFEFE;
const uint64_t MACROmask3 = 0x007F7F7F7F7F7F7F;
const uint64_t MACROmask4 = 0xFEFEFEFEFEFEFE00;
const uint64_t MACROmask5 = 0xFFFFFFFFFFFFFFFF;
const uint64_t MACROmask6 = 0xFFFFFFFFFFFFFFFF;
const uint64_t MACROmask7 = 0x00FEFEFEFEFEFEFE;
const uint64_t MACROmask8 = 0x7F7F7F7F7F7F7F00;


__device__ void getAllLegalMoves_d(int8_t ** mlPointer, uint64_t friendlyStones, uint64_t enemyStones, bool turn) {
    // A temporary holder for the moves in each direction
    uint64_t tempMoves;
    // Squares that don't have a stone on them.
    uint64_t emptySquares = ~(friendlyStones | enemyStones);
    uint64_t output = 0;

    uint64_t fastMask;

    // Each set is 24 ASM instructions
    fastMask = MACROmask1 & enemyStones;
    tempMoves = (friendlyStones >> shift1 & fastMask);
    tempMoves |= (tempMoves >> shift1 & fastMask);
    tempMoves |= (tempMoves >> shift1 & fastMask);
    tempMoves |= (tempMoves >> shift1 & fastMask);
    tempMoves |= (tempMoves >> shift1 & fastMask);
    tempMoves |= (tempMoves >> shift1 & fastMask);
    output |= (tempMoves >> shift1 & MACROmask1) & emptySquares;

    fastMask = MACROmask2 & enemyStones;
    tempMoves = (friendlyStones << shift1 & fastMask);
    tempMoves |= (tempMoves << shift1 & fastMask);
    tempMoves |= (tempMoves << shift1 & fastMask);
    tempMoves |= (tempMoves << shift1 & fastMask);
    tempMoves |= (tempMoves << shift1 & fastMask);
    tempMoves |= (tempMoves << shift1 & fastMask);
    output |= (tempMoves << shift1 & MACROmask2) & emptySquares;

    fastMask = MACROmask3 & enemyStones;
    tempMoves = (friendlyStones >> shift2 & fastMask);
    tempMoves |= (tempMoves >> shift2 & fastMask);
    tempMoves |= (tempMoves >> shift2 & fastMask);
    tempMoves |= (tempMoves >> shift2 & fastMask);
    tempMoves |= (tempMoves >> shift2 & fastMask);
    tempMoves |= (tempMoves >> shift2 & fastMask);
    output |= (tempMoves >> shift2 & MACROmask3) & emptySquares;

    fastMask = MACROmask4 & enemyStones;
    tempMoves = (friendlyStones << shift2 & fastMask);
    tempMoves |= (tempMoves << shift2 & fastMask);
    tempMoves |= (tempMoves << shift2 & fastMask);
    tempMoves |= (tempMoves << shift2 & fastMask);
    tempMoves |= (tempMoves << shift2 & fastMask);
    tempMoves |= (tempMoves << shift2 & fastMask);
    output |= (tempMoves << shift2 & MACROmask4) & emptySquares;

    fastMask = MACROmask5 & enemyStones;
    tempMoves = (friendlyStones >> shift3 & fastMask);
    tempMoves |= (tempMoves >> shift3 & fastMask);
    tempMoves |= (tempMoves >> shift3 & fastMask);
    tempMoves |= (tempMoves >> shift3 & fastMask);
    tempMoves |= (tempMoves >> shift3 & fastMask);
    tempMoves |= (tempMoves >> shift3 & fastMask);
    output |= (tempMoves >> shift3 & MACROmask5) & emptySquares;

    fastMask = MACROmask6 & enemyStones;
    tempMoves = (friendlyStones << shift3 & fastMask);
    tempMoves |= (tempMoves << shift3 & fastMask);
    tempMoves |= (tempMoves << shift3 & fastMask);
    tempMoves |= (tempMoves << shift3 & fastMask);
    tempMoves |= (tempMoves << shift3 & fastMask);
    tempMoves |= (tempMoves << shift3 & fastMask);
    output |= (tempMoves << shift3 & MACROmask6) & emptySquares;

    fastMask = MACROmask7 & enemyStones;
    tempMoves = (friendlyStones >> shift4 & fastMask);
    tempMoves |= (tempMoves >> shift4 & fastMask);
    tempMoves |= (tempMoves >> shift4 & fastMask);
    tempMoves |= (tempMoves >> shift4 & fastMask);
    tempMoves |= (tempMoves >> shift4 & fastMask);
    tempMoves |= (tempMoves >> shift4 & fastMask);
    output |= (tempMoves >> shift4 & MACROmask7) & emptySquares;

    fastMask = MACROmask8 & enemyStones;
    tempMoves = (friendlyStones << shift4 & fastMask);
    tempMoves |= (tempMoves << shift4 & fastMask);
    tempMoves |= (tempMoves << shift4 & fastMask);
    tempMoves |= (tempMoves << shift4 & fastMask);
    tempMoves |= (tempMoves << shift4 & fastMask);
    tempMoves |= (tempMoves << shift4 & fastMask);
    output |= (tempMoves << shift4 & MACROmask8) & emptySquares;

    if(output) {
        // This little trick saves a few cycles over looping from 0 to 64.
        // Worst case is the amount of squares
        while(output) {
            *(*mlPointer)++ = __ffs(output);
            // output ^= ONE64 << __builtin_ctzl(output);
            output = (output-1) & output; // Slightly faster than the one above
        }
        return;
    }
    *(*mlPointer)++ = -1;
}

__device__ void turnStonesFromMove_d(int8_t square, uint64_t * friendlyStones, uint64_t * enemyStones) {
    // This assumes the piece is already placed, it will not
    // placed the piece or turn the pieces.
    uint64_t tempOutput;
    uint64_t piecePlaced = ONE64GPU << square;
    uint64_t ifCaptured;
    uint64_t output = 0;
    uint64_t fastMask;

    fastMask = MACROmask1 & *enemyStones;
    tempOutput = (piecePlaced >> shift1 & fastMask);
    tempOutput |= (tempOutput >> shift1 & fastMask);
    tempOutput |= (tempOutput >> shift1 & fastMask);
    tempOutput |= (tempOutput >> shift1 & fastMask);
    tempOutput |= (tempOutput >> shift1 & fastMask);
    tempOutput |= (tempOutput >> shift1 & fastMask);
    ifCaptured = (tempOutput >> shift1 & MACROmask1) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask2 & *enemyStones;
    tempOutput = (piecePlaced << shift1 & fastMask);
    tempOutput |= (tempOutput << shift1 & fastMask);
    tempOutput |= (tempOutput << shift1 & fastMask);
    tempOutput |= (tempOutput << shift1 & fastMask);
    tempOutput |= (tempOutput << shift1 & fastMask);
    tempOutput |= (tempOutput << shift1 & fastMask);
    ifCaptured = (tempOutput << shift1 & MACROmask2) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask3 & *enemyStones;
    tempOutput = (piecePlaced >> shift2 & fastMask);
    tempOutput |= (tempOutput >> shift2 & fastMask);
    tempOutput |= (tempOutput >> shift2 & fastMask);
    tempOutput |= (tempOutput >> shift2 & fastMask);
    tempOutput |= (tempOutput >> shift2 & fastMask);
    tempOutput |= (tempOutput >> shift2 & fastMask);
    ifCaptured = (tempOutput >> shift2 & MACROmask3) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask4 & *enemyStones;
    tempOutput = (piecePlaced << shift2 & fastMask);
    tempOutput |= (tempOutput << shift2 & fastMask);
    tempOutput |= (tempOutput << shift2 & fastMask);
    tempOutput |= (tempOutput << shift2 & fastMask);
    tempOutput |= (tempOutput << shift2 & fastMask);
    tempOutput |= (tempOutput << shift2 & fastMask);
    ifCaptured = (tempOutput << shift2 & MACROmask4) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask5 & *enemyStones;
    tempOutput = (piecePlaced >> shift3 & fastMask);
    tempOutput |= (tempOutput >> shift3 & fastMask);
    tempOutput |= (tempOutput >> shift3 & fastMask);
    tempOutput |= (tempOutput >> shift3 & fastMask);
    tempOutput |= (tempOutput >> shift3 & fastMask);
    tempOutput |= (tempOutput >> shift3 & fastMask);
    ifCaptured = (tempOutput >> shift3 & MACROmask5) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask6 & *enemyStones;
    tempOutput = (piecePlaced << shift3 & fastMask);
    tempOutput |= (tempOutput << shift3 & fastMask);
    tempOutput |= (tempOutput << shift3 & fastMask);
    tempOutput |= (tempOutput << shift3 & fastMask);
    tempOutput |= (tempOutput << shift3 & fastMask);
    tempOutput |= (tempOutput << shift3 & fastMask);
    ifCaptured = (tempOutput << shift3 & MACROmask6) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask7 & *enemyStones;
    tempOutput = (piecePlaced >> shift4 & fastMask);
    tempOutput |= (tempOutput >> shift4 & fastMask);
    tempOutput |= (tempOutput >> shift4 & fastMask);
    tempOutput |= (tempOutput >> shift4 & fastMask);
    tempOutput |= (tempOutput >> shift4 & fastMask);
    tempOutput |= (tempOutput >> shift4 & fastMask);
    ifCaptured = (tempOutput >> shift4 & MACROmask7) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    fastMask = MACROmask8 & *enemyStones;
    tempOutput = (piecePlaced << shift4 & fastMask);
    tempOutput |= (tempOutput << shift4 & fastMask);
    tempOutput |= (tempOutput << shift4 & fastMask);
    tempOutput |= (tempOutput << shift4 & fastMask);
    tempOutput |= (tempOutput << shift4 & fastMask);
    tempOutput |= (tempOutput << shift4 & fastMask);
    ifCaptured = (tempOutput << shift4 & MACROmask8) & *friendlyStones;
    output |= (ifCaptured ? tempOutput : 0);

    *friendlyStones ^= output;
    *enemyStones ^= output;
}


__device__ bool doMove_d(int8_t square, uint64_t * friendlyStones, uint64_t * enemyStones, bool * lastMoveSkipped, bool * turn) {
    // If the player is passing
    if(square == -1) {
        // if(pos.lastMoveSkipped) {
        //     return true;
        // }
        // pos.lastMoveSkipped = true;
        // pos.turn = !pos.turn;
        // return false;

        // The code above is what this is doing, but this is branchless
        *turn = !*turn;
        return !(*lastMoveSkipped = !*lastMoveSkipped);
    }

    turnStonesFromMove_d(square, friendlyStones, enemyStones);

    *friendlyStones |= ((ONE64GPU & *turn) << square);
    *enemyStones |= ((ONE64GPU & !*turn) << square);

    // Set the turn to the other player
    *turn = !*turn;

    // This move was not passed.
    *lastMoveSkipped = false;

    // The game is not over
    return false;
}

__device__ void doPerft_d(uint64_t team[2], bool lastMoveSkipped, bool turn, int32_t depth, uint64_t * output) {
    int8_t moveList[64];
    int8_t * last = &moveList[0];
    uint64_t teamLocal[2] = {team[0], team[1]};

    getAllLegalMoves_d(&last, teamLocal[turn], teamLocal[!turn], turn);
    if(depth == 1) {
        (*output) += last - moveList;
        return;
    }

    // No need to undo because these are passed by value.
    for(int i = 0; i < (last - moveList); i++) {
        if(doMove_d(moveList[i], &teamLocal[turn], &teamLocal[!turn], &lastMoveSkipped, &turn)) {
            (*output)++;
            return;
        }
        doPerft_d(teamLocal, lastMoveSkipped, turn, depth-1, output);
    }
}

__global__ void doPerftsOnGPU(uint64_t ** teams, bool * lastMovesSkipped, bool * turns, int32_t depth, uint64_t * outputs, uint64_t threads) {
    const int threadNumber = blockDim.x * blockIdx.x + threadIdx.x;
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadNumber);

    if(threadNumber < threads) {
        
        doPerft_d(teams[threadNumber], lastMovesSkipped[threadNumber], turns[threadNumber], depth, &outputs[threadNumber]);
    }
}